#include "hip/hip_runtime.h"
//==============================================================================
// Name        : cuda2.cpp
// Author      : Hendrik Thole & Christian Claus
// Version     :
// Copyright   : GNU Generel Public License v2 or later
// Compile     : Use the nvcc compiler
//               nvcc -O2 cuda2.cu -o calc
//
// Ressources  : The ressources folder has plain text files. Those contain
//				 numbers used to calculate the prefix sum.
//				 Valid format: "12 23 34 4556 0 1234" etc.
//==============================================================================

#include <stdio.h>
#include <hip/hip_runtime.h>

#define FILE_PATH		"./resources/"
#define FILE_MODE		"r"
#define TOKEN 			" "
#define BUF_SIZE		500000
#define EXIT_SUCCESS	0
#define EXIT_FAILURE	1


#define MAX_THREADS 	512

//==============================================================================

__global__ void prefix_block_sum(float *array, int elements);
__global__ void prefix_overall_sum(float *array, int elements);
__global__ void prefix_finalize(float *array, int elements);
__global__ void prefix_finalize_to_x(float *array, int elements);
__device__ __host__ long oplus (const long a, const long b);

void read_file(float[], int, char[]);
void checkCUDAError(const char *msg);

//==============================================================================

int main(int argc, char **argv) {

	if (argc != 4) {
		printf("usage: %s <device id> <size> <filename>\n", argv[0]);
		exit(1);
	}
	
	//--------------------------------------------------------------------------

	/* which device do we want to use? */
	int device = atoi(argv[1]);
	hipSetDevice(device);
	
	//--------------------------------------------------------------------------

	/* we need arrays on host and device */
	float *array_host, *array_device;

	const int elements = atoi(argv[2]);


	int threads 	   = MAX_THREADS; // or change to whatever you want
	int sub_blocks	   = 10;
	int blocks  	   = elements / threads / sub_blocks;
	int offset		   = elements % threads;
	
	if (offset != 0) {
		blocks++;	
	}
	
	if (blocks == 0) {
		blocks = 1;
	}
	
	printf("blocks:  %d\n", blocks);
	printf("threads: %d\n", threads);
	
	
	/* 
	 * Array für Host und Device allokieren
	 */
	
	size_t size 		 = elements * sizeof(float);
	array_host  = (float *) malloc(size);
	hipMalloc((void **) &array_device, size);

	if (array_host == NULL) {
		printf("doh! not enough memory...\n");
		
		return EXIT_FAILURE;
	}
	
	//--------------------------------------------------------------------------

	/* let's read the file into the host's array */
	read_file(array_host, elements, argv[3]);


	//--------------------------------------------------------------------------
	
	hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
	
	prefix_block_sum <<< blocks, threads >>> (array_device, elements);
	hipDeviceSynchronize();
	checkCUDAError("kernel invocation");
	
	prefix_overall_sum <<< 1, 1 >>> (array_device, elements);
	hipDeviceSynchronize();
	checkCUDAError("kernel invocation");
	
	prefix_finalize <<< blocks, threads >>> (array_device, elements);
	hipDeviceSynchronize();
	checkCUDAError("kernel invocation");
	
	prefix_finalize_to_x <<< 1, threads >>> (array_device, elements);
	hipDeviceSynchronize();
	checkCUDAError("kernel invocation");
	
	
	hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
	
	printf("x = %f\n", array_host[0]);
	
	//--------------------------------------------------------------------------
	
	// cleanup
	free(array_host);
	hipFree(array_device);
	
	return EXIT_SUCCESS;
}

//==============================================================================

/* each thread calcs a prefix sum in it's own */
__global__ void prefix_block_sum(float *array, int elements) {
	int uid   		 = blockIdx.x * blockDim.x + threadIdx.x;
	int start 		 = uid * 10;
	
	if (start < elements) {
		int end = start + 10;
		long sum = 0;
		
		for (int i = start; i < end && i < elements; i++) {
			sum = oplus(sum, array[i]);
			array[i] = sum;
		}
	}
	
}

//==============================================================================

/* prefix sum over the last element each thread calc'ed is generated */
__global__ void prefix_overall_sum(float *array, int elements) {
	long sum = 0;
	
	for (int i = 10; i < elements; i += 10) {
		sum  = oplus(sum, array[i - 1]);
		array[i - 1] = sum;
	}
}

//==============================================================================

/* now we let each thread add those to 'his' block of numbers */
__global__ void prefix_finalize(float *array, int elements) {
	unsigned int uid   = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int start = 10 * (uid + 1);

	if (start < elements) {
		unsigned int end = start + 10;

		for (unsigned int i = start; i < end - 1 && i < elements; i++) {
			array[i] = oplus(array[i], array[start - 1]);
		}
	}
}

//==============================================================================

/* calculate the final x */
__global__ void prefix_finalize_to_x(float *array, int elements) {
	
	unsigned int tx = threadIdx.x;
	unsigned int dim = blockDim.x;

	__shared__ float sum[MAX_THREADS];

	sum[tx] = 0;


	unsigned int add_parts = elements / dim;

	unsigned int arr_offset = elements%dim;

	// offset
	if (arr_offset != 0 && tx < arr_offset) {
		sum[tx] += array[dim * add_parts + tx];
	}

	for (unsigned int i = 0; i < add_parts; i++) {
		sum[tx] += array[tx * add_parts + i];
	}
	
	__syncthreads();
	
	// reduce
	for(unsigned int offset=dim>>1; offset>0; offset = offset >>1) {
		if(tx < offset) {
			sum[tx] += sum[tx + offset];
		}
		
		__syncthreads(); 
	} 
	
	if (tx == 0) {
		array[tx] = sum[tx];
	}
	
}
//==============================================================================

/* the function can be changed here, usually prefix *sums* are '+' ;-)
 * this may just give some flexibility
 */
__device__ __host__ long oplus (const long a, const long b) {
	return a + b;
}

//==============================================================================

void read_file(float list[], int count, char *filename) {
	FILE *file;
	char line[BUF_SIZE];
	char *z;
	char *abs_filename = (char *) malloc(strlen(filename) + strlen(FILE_PATH)
			+ 1);
	long cols = 0;

	strcpy(abs_filename, FILE_PATH);
	strcat(abs_filename, filename);

	file = fopen(abs_filename, "r");

	if (file == NULL) {
		printf("file %s could not be opened.\n", abs_filename);
	} else {
		while (fgets(line, sizeof(line), file) != NULL) {
			z = strtok(line, TOKEN);

			while (z != NULL && cols < count) {
				list[cols] = atoi(z);
				z = strtok(NULL, TOKEN);
				cols++;
			}
		}
	}
	free(abs_filename);

	fclose(file);
}

//==============================================================================

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    
    if(hipSuccess != err) {
        fprintf(stderr, "CUDA error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
