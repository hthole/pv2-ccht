#include "hip/hip_runtime.h"
/*
 *
 *      Author: C. Claus, H. Thole
 *
 *      Kompilieren mit:
 *      nvcc -O2 -g -deviceemu cuda.cu -o cuda
 *
 *      oder
 *
 *      nvcc -O2 -g cuda.cu -o cuda
 *
 *
 *      Ausführen mit:
 *      ./cuda  <Device ID> <Anzahl der Zahlen> <Dateiname>
 *
 *      Die einzulesenden Dateien muessen sich in "../resources/" befinden.
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define FILE_PATH	"./resources/"
#define FILE_MODE	"r"
#define TOKEN 		" "
#define BUF_SIZE	500000


__global__ void prefix_sum(int *array, float *all_sums_device);
__global__ void prefix_all_sum(int *array, float *all_sums_device) ;
__device__ __host__ long oplus (const long a, const long b);
void read_file(int[], int, char[]);
void print_debug(int list[], int tmp[], int count);
void checkCUDAError(const char *msg);






int main(int argc, char **argv) {
	/* Parameteranzahl pruefen */
	if (argc != 4) {
		printf("usage: %s <device id> <size> <filename>\n", argv[0]);
		exit(1);
	}

	//double start_time, end_time; // fuer Zeitnahme

	/* Device setzen */
	hipSetDevice (atoi(argv[1]));

	/*  Host- und Devicearray einrichten */
	int *array_host, *array_host_seq, *array_device;
	float *all_sums, *all_sums_device;

	// konvertiere die Uebergabeparameter von char zu int
	const int elements = atoi(argv[2]);

	int j = 0;
	while (ldexp(1,j) < elements) {
		j++;
		printf("%d ",j);
	}

	if (ldexp(1,j) == elements) {
		printf("zweierpotenz!\n");
	} else {
		printf("\n%d -- keine zweierpotenz!\n", j);
	}

	/* Array für Host und Device allokieren */
	size_t size = elements * sizeof(int);
	array_host = (int *) malloc(size);
	array_host_seq = (int *) malloc(size);
	hipMalloc((void **) &array_device, size);

	if (array_host == NULL) {
		printf("Nicht genug Speicher...\n");
		exit(0);
	}
	if (array_host_seq == NULL) {
		printf("Nicht genug Speicher...\n");
		exit(0);
	}


	/* Datei in Hostarray einlesen */
	read_file(array_host, elements, argv[3]);
	read_file(array_host_seq, elements, argv[3]);

	/* Hostarray zu CUDA-Device kopieren */
	hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);

	int i=0;
	for (i = 1; i < elements; i++) {
		array_host_seq[i] = array_host_seq[i - 1] + array_host_seq[i];
	}

	int ergebnis = 0;
	for (i = 0; i < elements; i++) {
		ergebnis += array_host_seq[i];
	}

	// just for developing block-stuff
	int num_blocks = 2;
	int num_threads_per_block = elements / (2* num_blocks);

	// all_sums array einrichten... ein feld pro block
	size_t size_all_sums = num_blocks * sizeof(float);
	all_sums = (float *) malloc(size_all_sums);
	hipMalloc((void **) &all_sums_device, size_all_sums);

	// und rueber damit...
	//hipMemcpy(all_sums_device, all_sums, size_all_sums, hipMemcpyHostToDevice);

//	const int max_threads = 512; // max threads pro block
//	const int num_blocks = elements / max_threads; // anzahl der blocks
//	const int num_threads_per_block = max_threads/2; // anzahl der tatsaechlichen threads pro block

	// Do calculation on device:
	prefix_sum <<< num_blocks, num_threads_per_block >>> (array_device, all_sums_device);
	// Retrieve result from device and store it in host array
	//hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);

	// block until the device has completed
	    hipDeviceSynchronize();
	    checkCUDAError("kernel invocation");



	hipMemcpy(all_sums, all_sums_device, size_all_sums, hipMemcpyDeviceToHost);

	for (int m = 1; m <= num_blocks; m++) {
		printf("all_sums: %f\n", all_sums[m-1]);
	}

	// zwischensummen berechnen
	for (int m = 1; m <= num_blocks; m++) {
		all_sums[m] = all_sums[m - 1] + all_sums[m];
	}

	// auf dem device weiterrechnen
	hipMemcpy(all_sums_device, all_sums, size_all_sums, hipMemcpyHostToDevice);
	prefix_all_sum <<< num_blocks, num_threads_per_block >>> (array_device, all_sums_device);

	// block until the device has completed
	    hipDeviceSynchronize();
	    checkCUDAError("kernel invocation");



	hipMemcpy(all_sums, all_sums_device, size_all_sums, hipMemcpyDeviceToHost);

	for (int m = 1; m <= num_blocks; m++) {
		printf("ende all_sums: %f\n", all_sums[m-1]);
	}

	float overall_sum = 0;
	for (int v = 0; v<num_blocks; v++) {
		overall_sum += all_sums[v];
	}







	// Print results
	//print_debug(array_host, array_host_seq, elements);

	printf ("%f <--- korrekt waere %i \n", overall_sum, ergebnis);

	for (int i = 0; i < elements; i++) {
		//assert (array_host[i] == array_host_seq[i]);
	}

	// Cleanup
	free(array_host);
	free(array_host_seq);
	hipFree(array_device);
}



/* Unser Kernel */
__global__ void prefix_sum(int *array, float *all_sums_device) {

	// shared temp array
	extern __shared__ int temp[];

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int dim = blockDim.x;

	int offset = 1;

	// schreibe das array in shared speicher
	temp[2 * tx] = array[2 * tx + (bx*dim*2)];
	temp[2 * tx + 1] = array[2 * tx + 1 + (bx*dim*2)];

	// teilsummen bilden
	for (int steps = dim; steps > 0; steps /= 2) {
		__syncthreads();

		if (tx < steps) {
			int first = offset * (2 * tx + 1) - 1;
			int second = offset * (2 * tx + 2) - 1;

			temp[second] += temp[first];
		}

		offset *= 2;
	}

	//
	//
	//

	__syncthreads();

	// letztes element sichern, dann nullen, damit das ergebnis korrekt wird
	int tmp_var = -1;
	if (tx == 0) {
		tmp_var = temp[(2 * dim) - 1];
		all_sums_device[bx] = tmp_var;
		temp[(2 * dim) - 1] = 0;
		//printf("var oben: %i\n", tmp_var);
		//printf("all_sums_device: %f\n", all_sums_device[bx]);
	}

	__syncthreads();

	// werte durchtauschen
	for (int steps = 1; steps <= dim; steps *= 2) {
		offset /= 2;
		__syncthreads(); //<-------------------------------- unnoetig??????

		if (tx < steps) {
			int first = offset * (2 * tx + 1) - 1;
			int second = offset * (2 * tx + 2) - 1;

			int t = temp[first];
			temp[first] = temp[second];
			temp[second] += t;
		}
	}



	__syncthreads(); //<-------------------------------- unnoetig??????

	// ins array zurückschreiben
	array[2 * tx + (bx*dim)] = temp[2 * tx];
	array[2 * tx + 1 + (bx*dim)] = temp[2 * tx + 1];

	__syncthreads();

	if (tx == 0) {
		//printf("stelle im array: %i\n", ntpb-1 + (bx*ntpb));
		//printf("var unten: %i\n", tmp_var);
		array[bx * ((2 * dim) - 1)] = tmp_var;
	}

	//printf("thread %i in block % i -- %i\n", tx, bx, array[2 * tx + (bx*dim)]);
	//printf("thread %i in block % i -- %i\n", tx, bx, array[2 * tx + 1 + (bx*dim)]);
	__syncthreads();
}

__global__ void prefix_all_sum(int *array, float *all_sums_device) {

	// shared temp array
	extern __shared__ int temp[];


	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int dim = blockDim.x;

	int offset = 1;

	// schreibe das array in shared speicher
	temp[2 * tx] = array[2 * tx + (bx*dim*2)];
	temp[2 * tx + 1] = array[2 * tx + 1 + (bx*dim*2)];

	__syncthreads();

	// addiere zwischensummen auf
	if (bx > 0) {
		temp[2 * tx] += all_sums_device[bx - 1];
		temp[2 * tx + 1] += all_sums_device[bx - 1];
	}

	__syncthreads();

	/*
	 * wir haben jetzt korrekte teilsummen... nun wieder obersumme im threadblock bilden, in all_sums schreiben,
	 * zurück zum host und aufaddieren
	 */

	// teilsummen bilden
	for (int steps = dim; steps > 0; steps /= 2) {
		__syncthreads();

		if (tx < steps) {
			int first = offset * (2 * tx + 1) - 1;
			int second = offset * (2 * tx + 2) - 1;

			temp[second] += temp[first];
		}

		offset *= 2;
	}

	__syncthreads();

	if (tx == 0) {
		all_sums_device[bx] = temp[(dim*2) - 1];
		//printf("ende all_sums_device: %f\n", all_sums_device[bx]);
	}


}


/* Eine tolle assoziative Device-Funktion ... */
__device__ __host__ long oplus (const long a, const long b) {
	return a + b;
}



void read_file(int list[], int count, char *filename) {
	FILE *file;
	char line[BUF_SIZE];
	char *z;
	char *abs_filename = (char *) malloc(strlen(filename) + strlen(FILE_PATH)
			+ 1);
	long cols = 0;

	strcpy(abs_filename, FILE_PATH);
	strcat(abs_filename, filename);

	file = fopen(abs_filename, "r");

	if (file == NULL) {
		printf("Datei %s konnte nicht geoeffnet werden.\n", abs_filename);
	} else {
		while (fgets(line, sizeof(line), file) != NULL) {
			// fuer jede zeile
			z = strtok(line, TOKEN);

			while (z != NULL && cols < count) {
				// fuer jede spalte (zahl) in der zeile
				list[cols] = atoi(z);
				z = strtok(NULL, TOKEN);
				cols++;
			}
		}
	}
	free(abs_filename);

	fclose(file);
}

void print_debug(int list[], int tmp[], int count) {
	int i;

	for (i = 0; i < count; i++) {
		fprintf(stderr, "%i. %i sollte sein %i\n", i, list[i], tmp[i]);
	}
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

