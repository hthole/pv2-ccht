/*
 *
 *      Author: C. Claus, H. Thole
 *
 *      Kompilieren mit:
 *      nvcc -O2 -g -deviceemu cuda.cu -o cuda
 *
 *      oder
 *
 *      nvcc -O2 -g cuda.cu -o cuda
 *
 *
 *      Ausführen mit:
 *      ./cuda  <Device ID> <Anzahl der Zahlen> <Dateiname>
 *
 *      Die einzulesenden Dateien muessen sich in "../resources/" befinden.
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define FILE_PATH	"./resources/"
#define FILE_MODE	"r"
#define TOKEN 		" "
#define BUF_SIZE	500000


__global__ void prefix_sum(int *array, int *tmp_array, int elements);
__device__ __host__ long oplus (const long a, const long b);
void read_file(int[], int, char[]);
void print_debug(int list[], int tmp[], int count);






int main(int argc, char **argv) {
	/* Parameteranzahl pruefen */
	if (argc != 4) {
		printf("usage: %s <device id> <size> <filename>\n", argv[0]);
		exit(1);
	}

	//double start_time, end_time; // fuer Zeitnahme

	/* Device setzen */
	hipSetDevice (atoi(argv[1]));

	/*  Host- und Devicearray einrichten */
	int *array_host, *array_host_tmp, *array_device, *array_device_tmp;

	// konvertiere die Uebergabeparameter von char zu int
	const int elements = atoi(argv[2]);

	/* Array für Host und Device allokieren */
	size_t size = elements * sizeof(int);
	array_host = (int *) malloc(size);
	array_host_tmp = (int *) malloc(size);
	hipMalloc((void **) &array_device, size);
	hipMalloc((void **) &array_device_tmp, size);


	/* Datei in Hostarray einlesen */
	read_file(array_host, elements, argv[3]);
	read_file(array_host_tmp, elements, argv[3]);

	/* Hostarray zu CUDA-Device kopieren */
	hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
	hipMemcpy(array_device_tmp, array_host, size, hipMemcpyHostToDevice);

	int i=0;
	for (i = 1; i < elements; i++) {
		array_host_tmp[i] = array_host_tmp[i - 1] + array_host_tmp[i];
	}



	// Do calculation on device:
	prefix_sum <<< 1, elements >>> (array_device, array_device_tmp, elements);
	// Retrieve result from device and store it in host array
	hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);

	// Print results
	//print_debug(array_host, array_host_tmp, elements);

	for (int i = 0; i < elements; i++) {
		assert (array_host[i] == array_host_tmp[i]);
	}

	// Cleanup
	free(array_host);
	hipFree(array_device);
}



/* Unser Kernel */
__global__ void prefix_sum(int *array, int *array_tmp, int elements) {

	// Dynamically allocated shared memory for scan kernels
	extern __shared__ float temp[];

	int thid = threadIdx.x;

	int offset = 1;

	// Cache the computational window in shared memory
	temp[2 * thid] = array_tmp[2 * thid];
	temp[2 * thid + 1] = array_tmp[2 * thid + 1];

	// build the sum in place up the tree
	for (int steps = elements/2; steps > 0; steps /= 2) {
		__syncthreads();

		if (thid < steps) {
			int first = offset * (2 * thid + 1) - 1;
			int second = offset * (2 * thid + 2) - 1;

			temp[second] += temp[first];
		}

		offset *= 2;
	}

	// scan back down the tree

	// clear the last element
	int tmp_var = 0;
	if (thid == 0) {
		tmp_var = temp[elements - 1];
		temp[elements - 1] = 0;
	}

	// traverse down the tree building the scan in place
	for (int steps = 1; steps < elements; steps *= 2) {
		offset /= 2;
		//__syncthreads(); <-------------------------------- unnoetig??????

		if (thid < steps) {
			int first = offset * (2 * thid + 1) - 1;
			int second = offset * (2 * thid + 2) - 1;

			float t = temp[first];
			temp[first] = temp[second];
			temp[second] += t;
		}
	}



	__syncthreads();

	// write results to global memory
	array[(2 * thid) - 1] = temp[2 * thid];
	array[2 * thid] = temp[2 * thid + 1];

	__syncthreads();

	if (thid == 0) {
			array[elements-1] = tmp_var;
	}


}


/* Eine tolle assoziative Device-Funktion ... */
__device__ __host__ long oplus (const long a, const long b) {
	return a + b;
}



void read_file(int list[], int count, char *filename) {
	FILE *file;
	char line[BUF_SIZE];
	char *z;
	char *abs_filename = (char *) malloc(strlen(filename) + strlen(FILE_PATH)
			+ 1);
	long cols = 0;

	strcpy(abs_filename, FILE_PATH);
	strcat(abs_filename, filename);

	file = fopen(abs_filename, "r");

	if (file == NULL) {
		printf("Datei %s konnte nicht geoeffnet werden.\n", abs_filename);
	} else {
		while (fgets(line, sizeof(line), file) != NULL) {
			// fuer jede zeile
			z = strtok(line, TOKEN);

			while (z != NULL && cols < count) {
				// fuer jede spalte (zahl) in der zeile
				list[cols] = atoi(z);
				z = strtok(NULL, TOKEN);
				cols++;
			}
		}
	}
	free(abs_filename);

	fclose(file);
}

void print_debug(int list[], int tmp[], int count) {
	int i;

	for (i = 0; i < count; i++) {
		fprintf(stderr, "%i. %i sollte sein %i\n", i, list[i], tmp[i]);
	}
}
